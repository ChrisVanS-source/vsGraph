#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "qnvidia_backward.cuh"
#include "qgraph_gpu.h"

template void run_backward_nvidia(int device, const graph_gpu_memory<double>& graph_mem, workspace_gpu_memory<double>& workspace_mem, ADType direction);
template void run_backward_nvidia(int device, const graph_gpu_memory<float>& graph_mem, workspace_gpu_memory<float>& workspace_mem, ADType direction);

#include "qgraph_backward_gpu.h"
#include "qgraph_function_gpu.h"

template<class T>
__global__ void run_backward_device(qgraph_gpu<T> g, graph_function_gpu<T> fwd, graph_backward_gpu<T> bkwd, size_t n, ADType direction)
{
	uint64_t threadID = blockDim.x * blockIdx.x + threadIdx.x;

	if (threadID >= n) return;

	uint64_t skip_index = 0;

	if (direction != BACKWARD)
		g.sweep_forward(fwd, threadID);
	if (direction != FORWARD)
		g.sweep_backward(bkwd, fwd, skip_index, threadID);
}

#include "qexception.h"
using namespace vstech;

template<class T>
void run_backward_nvidia(int device, const graph_gpu_memory<T>& graph_mem, workspace_gpu_memory<T>& workspace_mem, ADType direction)
{
	hipSetDevice(device);

	uint64_t n = workspace_mem.m_n;
	
	dim3 blockSize(128, 1, 1);
	dim3 gridSize((unsigned int)n / 128 + 1, 1);

	run_backward_device<<<gridSize, blockSize>>>(
		
		qgraph_gpu<T>(graph_mem), 
		graph_function_gpu<T>(graph_mem, workspace_mem),
		graph_backward_gpu<T>(graph_mem, workspace_mem),
		n,
		direction

	);

	hipError_t cudaStatus = hipGetLastError();

	if (cudaStatus != hipSuccess)
		throw qexception("Error running graph function on NVIDIA");

	const auto& wksp_index = workspace_mem.m_output_wksp_index;
	T* p = workspace_mem.m_hh_mem.get();

	for (uint64_t i = 0; i < wksp_index.size(); ++i)
	{
		hipMemcpy(p, workspace_mem.m_hh_wksp[wksp_index[i]], n * sizeof(T), hipMemcpyDeviceToHost);
		p += n;
	}
}

