#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "qnvidia_utils.cuh"

int get_num_nvidia()
{
    int nDevices;
    if (hipSuccess != hipGetDeviceCount(&nDevices))
    	return 0;
    return nDevices;
}

__global__ void dummykernel_nvidia() {}

#include <vector>
#include <memory>

class nvidia_helper
{
    static std::shared_ptr<std::pair<std::vector<bool>, std::vector<int>>> m_fpo;

public:

    nvidia_helper() 
    {
        if (m_fpo.get() == 0)
        {
            m_fpo.reset(new std::pair<std::vector<bool>, std::vector<int>>());
            std::pair<std::vector<bool>, std::vector<int>>& fc = *m_fpo.get();
            for (int i = 0; i < get_num_nvidia(); ++i)
            {
                hipSetDevice(i);
                dummykernel_nvidia<<<1,1>>>();
                fc.first.push_back(hipGetLastError() == hipSuccess);
                if (fc.first[i])
                    fc.second.push_back(i);
            }
        }
    }

    ~nvidia_helper() {}
    
    int get_num_fpoc_nvidia() const 
    { 
       	return (int)m_fpo->second.size(); 	
    }

    int get_fpoc_nvidia_device(unsigned int n) const 
    {
    	return n < m_fpo->second.size() ? m_fpo->second[n] : -1;
    }

};

std::shared_ptr<std::pair<std::vector<bool>, std::vector<int>>> nvidia_helper::m_fpo;

int get_num_fpoc_nvidia()
{
    return nvidia_helper().get_num_fpoc_nvidia();
}

int get_fpoc_nvidia_device(unsigned int n)
{
    return nvidia_helper().get_fpoc_nvidia_device(n);
}



// Graph memory

template void allocate_nvidia<double>(int device,
    graph_gpu_memory<double>& memory,
    uint64_t var_counter,
    const std::vector<Operation>& ops,
    const std::vector<uint64_t>& ops_arg_index,
    const std::vector<std::vector<uint64_t>>& wksp_index,
    const std::vector<double>& scalar,
    const std::vector<int64_t>& integer,
    const std::vector<bool>& boolean,
    const std::vector<uint64_t>& skip_ops_init,
    const std::vector<uint64_t>& skip_ops_arg_init,
    const std::vector<uint64_t>& checkpoint_ops_index,
    const std::vector<uint64_t>& checkpoint_ops_arg_index);

template void allocate_nvidia<float>(int device,
    graph_gpu_memory<float>& memory,
    uint64_t var_counter,
    const std::vector<Operation>& ops,
    const std::vector<uint64_t>& ops_arg_index,
    const std::vector<std::vector<uint64_t>>& wksp_index,
    const std::vector<float>& scalar,
    const std::vector<int64_t>& integer,
    const std::vector<bool>& boolean,
    const std::vector<uint64_t>& skip_ops_init,
    const std::vector<uint64_t>& skip_ops_arg_init,
    const std::vector<uint64_t>& checkpoint_ops_index,
    const std::vector<uint64_t>& checkpoint_ops_arg_index);

// Workspace memory

template void allocate_nvidia<double>(uint64_t n, int device,
    workspace_gpu_memory<double>& memory,
    const std::vector<uint64_t>& output_wksp_index,
    std::vector<double*>& wksp,
    std::shared_ptr<double[]> mem,
    std::vector<std::vector<int64_t>>& integer_wksp,
    std::vector<std::vector<bool>>& boolean_wksp,
    const std::vector<uint64_t>& wksp_index,
    const std::vector<uint64_t>& integer_wksp_index,
    const std::vector<uint64_t>& boolean_wksp_index,
    const std::vector<std::map<uint64_t, uint64_t>>& assign_map,
    const std::vector<double>& scalar,
    const std::vector<int64_t>& integer,
    const std::vector<bool>& boolean);

template void allocate_nvidia<float>(uint64_t n, int device,
    workspace_gpu_memory<float>& memory,
    const std::vector<uint64_t>& output_wksp_index,
    std::vector<float*>& wksp,
    std::shared_ptr<float[]> mem,
    std::vector<std::vector<int64_t>>& integer_wksp,
    std::vector<std::vector<bool>>& boolean_wksp,
    const std::vector<uint64_t>& wksp_index,
    const std::vector<uint64_t>& integer_wksp_index,
    const std::vector<uint64_t>& boolean_wksp_index,
    const std::vector<std::map<uint64_t, uint64_t>>& assign_map,
    const std::vector<float>& scalar,
    const std::vector<int64_t>& integer,
    const std::vector<bool>& boolean);

hipError_t freeDeviceMemory(void* devPtr)
{
    return hipFree(devPtr);
}


template<class T>
hipError_t allocate_copy(std::shared_ptr<T[]>& vd, const std::vector<T>& vh)
{
    hipError_t cudaStatus;
    T* p = 0;
    cudaStatus = hipMalloc((void**)&p, vh.size() * sizeof(T));
    if (cudaStatus != hipSuccess)
        return cudaStatus;
    vd.reset(p, freeDeviceMemory);

    if (vh.size() == 0)
        return cudaStatus;

    cudaStatus = hipMemcpy(p, &vh[0], vh.size() * sizeof(T), hipMemcpyHostToDevice);
    return cudaStatus;
}

template<>
hipError_t allocate_copy<bool>(std::shared_ptr<bool[]>& vd, const std::vector<bool>& _vh)
{
    std::shared_ptr<bool[]> vh(new bool[_vh.size()]);

    for (uint64_t i = 0; i < _vh.size(); ++i)
        vh.get()[i] = _vh[i];

    hipError_t cudaStatus;
    bool* p = 0;
    cudaStatus = hipMalloc((void**)&p, _vh.size() * sizeof(bool));
    if (cudaStatus != hipSuccess)
        return cudaStatus;
    vd.reset(p, freeDeviceMemory);

    if (_vh.size() == 0)
        return cudaStatus;

    cudaStatus = hipMemcpy(p, vh.get(), _vh.size() * sizeof(bool), hipMemcpyHostToDevice);
    return cudaStatus;
}

#include "qexception.h"
using namespace vstech;

#define CUDA_CHECK(status) if (status != hipSuccess) throw qexception("Memory allocation/copy failed on NVIDIA GPU")

// Graph

template<class T>
void allocate_nvidia(int device,
    graph_gpu_memory<T>& memory,
    uint64_t var_counter,
    const std::vector<Operation>& ops,
    const std::vector<uint64_t>& ops_arg_index,
    const std::vector<std::vector<uint64_t>>& wksp_index,
    const std::vector<T>& scalar,
    const std::vector<int64_t>& integer,
    const std::vector<bool>& boolean,
    const std::vector<uint64_t>& skip_ops_init,
    const std::vector<uint64_t>& skip_ops_arg_init,
    const std::vector<uint64_t>& checkpoint_ops_index,
    const std::vector<uint64_t>& checkpoint_ops_arg_index)
{
    hipSetDevice(device);

    memory.m_n_ops = ops.size();
    memory.m_n_ops_arg_index = ops_arg_index.size();
    memory.m_var_counter = var_counter;
    memory.m_n_checkpoint_ops_index = checkpoint_ops_index.size();

    CUDA_CHECK(allocate_copy<Operation>(memory.m_ops, ops));
    CUDA_CHECK(allocate_copy<uint64_t>(memory.m_ops_arg_index, ops_arg_index));
    CUDA_CHECK(allocate_copy<uint64_t>(memory.m_wksp_index, wksp_index[QFLOATING_POINT]));
    CUDA_CHECK(allocate_copy<uint64_t>(memory.m_integer_wksp_index, wksp_index[QINTEGER]));
    CUDA_CHECK(allocate_copy<uint64_t>(memory.m_boolean_wksp_index, wksp_index[QBOOLEAN]));
    CUDA_CHECK(allocate_copy<T>(memory.m_scalar, scalar));
    CUDA_CHECK(allocate_copy<int64_t>(memory.m_integer, integer));
    CUDA_CHECK(allocate_copy<bool>(memory.m_boolean, boolean));
    CUDA_CHECK(allocate_copy<uint64_t>(memory.m_skip_ops_init, skip_ops_init));
    CUDA_CHECK(allocate_copy<uint64_t>(memory.m_skip_ops_arg_init, skip_ops_arg_init));
    CUDA_CHECK(allocate_copy<uint64_t>(memory.m_checkpoint_ops_index, checkpoint_ops_index));
    CUDA_CHECK(allocate_copy<uint64_t>(memory.m_checkpoint_ops_arg_index, checkpoint_ops_arg_index));
}

// Workspace

#include <thrust/fill.h>
#include <thrust/device_vector.h>

template<class T>
void assign_nvidia(T* x, const T& x0, size_t n)
{
    thrust::device_ptr<T> ptr(x);
    thrust::fill(ptr, ptr + n, x0);
}


template<class T>
void allocate_nvidia(uint64_t n, int device,
    workspace_gpu_memory<T>& memory,
    const std::vector<uint64_t>& output_wksp_index,
    std::vector<T*>& wksp,
    std::shared_ptr<T[]> mem,
    std::vector<std::vector<int64_t>>& integer_wksp,
    std::vector<std::vector<bool>>& boolean_wksp,
    const std::vector<uint64_t>& wksp_index,
    const std::vector<uint64_t>& integer_wksp_index,
    const std::vector<uint64_t>& boolean_wksp_index,
    const std::vector<std::map<uint64_t, uint64_t>>& assign_map,
    const std::vector<T>& scalar,
    const std::vector<int64_t>& integer,
    const std::vector<bool>& boolean)
{
    hipSetDevice(device);

    memory.m_n = n;
    memory.m_hh_mem = mem;
    memory.m_output_wksp_index = output_wksp_index;

    // Scalar

    T* p = 0;

    CUDA_CHECK(hipMalloc((void**)&p, n * wksp.size() * sizeof(T)));
    memory.m_mem_wksp.reset(p, freeDeviceMemory);

    memory.m_hh_wksp.resize(wksp.size());

    for (uint64_t i = 0; i < wksp.size(); ++i) {
        memory.m_hh_wksp[i] = p;
        p += n;
    }

    CUDA_CHECK(allocate_copy<T*>(memory.m_wksp, memory.m_hh_wksp));

    for (auto it = assign_map[QFLOATING_POINT].begin(); it != assign_map[QFLOATING_POINT].end(); ++it)
        assign_nvidia(memory.m_hh_wksp[wksp_index[it->first]], scalar[it->second], n);

    // Integer

    int64_t* integer_p = 0;

    CUDA_CHECK(hipMalloc((void**)&integer_p, n * integer_wksp.size() * sizeof(int64_t)));

    memory.m_mem_integer_wksp.reset(integer_p, freeDeviceMemory);

    memory.m_hh_integer_wksp.resize(integer_wksp.size());

    for (size_t i = 0; i < integer_wksp.size(); ++i) {
        memory.m_hh_integer_wksp[i] = integer_p;
        integer_p += n;
    }

    CUDA_CHECK(allocate_copy<int64_t*>(memory.m_integer_wksp, memory.m_hh_integer_wksp));

    for (auto it = assign_map[QINTEGER].begin(); it != assign_map[QINTEGER].end(); ++it)
        assign_nvidia(memory.m_hh_integer_wksp[integer_wksp_index[it->first]], integer[it->second], n);

    // Boolean

    bool* boolean_p = 0;

    CUDA_CHECK(hipMalloc((void**)&boolean_p, n * boolean_wksp.size() * sizeof(bool)));

    memory.m_mem_boolean_wksp.reset(boolean_p, freeDeviceMemory);

    memory.m_hh_boolean_wksp.resize(boolean_wksp.size());

    for (size_t i = 0; i < boolean_wksp.size(); ++i) {
        memory.m_hh_boolean_wksp[i] = boolean_p;
        boolean_p += n;
    }

    CUDA_CHECK(allocate_copy<bool*>(memory.m_boolean_wksp, memory.m_hh_boolean_wksp));

    for (auto it = assign_map[QBOOLEAN].begin(); it != assign_map[QBOOLEAN].end(); ++it)
        assign_nvidia(memory.m_hh_boolean_wksp[boolean_wksp_index[it->first]], boolean[it->second], n);
}

// Set workspace

template void set_wksp_nvidia(int device, workspace_gpu_memory<double>& memory, uint64_t x, const double& xv);
template void set_wksp_nvidia(int device, workspace_gpu_memory<float>& memory, uint64_t x, const float& xv);

template<class T>
void set_wksp_nvidia(int device, workspace_gpu_memory<T>& memory, uint64_t x, const T& xv)
{
    hipSetDevice(device);
    assign_nvidia(memory.m_hh_wksp[x], xv, memory.m_n);
}

template void set_wksp_nvidia(int device, workspace_gpu_memory<double>& memory, uint64_t x, const double* xv);
template void set_wksp_nvidia(int device, workspace_gpu_memory<float>& memory, uint64_t x, const float* xv);

template<class T>
void set_wksp_nvidia(int device, workspace_gpu_memory<T>& memory, uint64_t x, const T* xv)
{
    hipSetDevice(device);
    hipMemcpy(memory.m_hh_wksp[x], xv, memory.m_n * sizeof(T), hipMemcpyHostToDevice);
}
